#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void getmaxcu(unsigned int * num, unsigned int size, unsigned int * max)
{
//  printf("threadid %d blockid %d blockdim %d\n", threadIdx.x , blockIdx.x , blockDim.x);
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    int counter = size/blockDim.x;

    int start = index*counter;
    int stop = start+counter;
    unsigned int localMax = 0;

    for (int i = start ; i < stop ;i++) {
      if(num[i] > localMax) {
        localMax = num[i];
      }
    }
    max[index] = localMax;
}

//unsigned int getmax(unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array

    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;

    unsigned int totalSize = size* sizeof(unsigned int);
    unsigned int * num_d;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    int numThreads = prop.maxThreadsPerMultiProcessor;
    int numBlocks = numThreads/prop.maxThreadsPerBlock;
    int numThreadspBlock = numThreads/numBlocks;
    if(size < numThreadspBlock) {
      numBlocks = 1;
      numThreadspBlock = size;
      numThreads = size;
    }

    unsigned int * maxArray;
    maxArray = (unsigned int *)malloc(numThreads * sizeof(unsigned int));
    unsigned int * maxArray_d;

    hipMalloc((void **) &num_d, totalSize);
    hipMalloc((void **) &maxArray_d, numThreads*sizeof(unsigned int));
    hipMemcpy(num_d, numbers, totalSize, hipMemcpyHostToDevice);
    hipMemcpy(maxArray_d, maxArray, numThreads*sizeof(unsigned int), hipMemcpyHostToDevice);

    //getmaxcu<<<numBlocks,numThreadspBlock>>>(num_d, size, maxArray_d);
    getmaxcu<<<(numBlocks+numThreadspBlock-1)/numThreadspBlock,numThreadspBlock>>>(num_d, size, maxArray_d);
    hipMemcpy(maxArray, maxArray_d, numBlocks*numThreadspBlock*sizeof(unsigned int),hipMemcpyDeviceToHost);

    unsigned int max = 0 ;
    for (int i = 0 ; i < numBlocks*numThreadspBlock; i++) {
      if (maxArray[i] > max){
        max = maxArray[i];
      }
    }

    printf(" parallel %u\n", max);
//    printf(" serial %u\n", getmax(numbers, size));


    free(numbers);
    hipFree(maxArray_d);
    hipFree(num_d);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array


unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}
*/
